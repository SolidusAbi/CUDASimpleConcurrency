
#include <hip/hip_runtime.h>
#include <pthread.h>
#include <stdio.h>
#include <iostream>

//const int N = 1 << 20;
const int N = 10;

__global__ void kernel(float *x, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        x[i] = sqrt(pow(3.14159,i));
    }
}

__global__ void idxTest(float *x, float *data, int n, uint incr)
{
	size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
	size_t stride = blockDim.x * gridDim.x;
	while (tid < n)
	{
		//x[tid] = static_cast<float>(tid);
		x[tid] = data[tid] + incr;
		tid += stride;
	}
}

/*int main()
{
    const int num_streams = 8;

    cudaStream_t streams[num_streams];
    float *data[num_streams];

    for (int i = 0; i < num_streams; i++) {
        cudaStreamCreate(&streams[i]);

        cudaMalloc(&data[i], N * sizeof(float));

        // launch one worker kernel per stream
        kernel<<<1, 64, 0, streams[i]>>>(data[i], N);

        // launch a dummy kernel on the default stream
        kernel<<<1, 1>>>(0, 0);
    }

    cudaDeviceReset();

    return 0;
}*/

struct cuda_streams_arg {
	hipStream_t *stream;
    float *data;
    uint threadId;
};

void *launch_kernel(void *args)
{

	cuda_streams_arg *thread_arg = static_cast<cuda_streams_arg *>(args);

	hipStream_t *currentStream = static_cast<hipStream_t *>(thread_arg->stream);
	hipStreamCreate(currentStream);

    float *data;
    hipMalloc(&data, N * sizeof(float));

    //kernel<<<1, 64, 0, *currentStream>>>(data, thread_arg->data, N);
    idxTest<<<2, 4, 0, *currentStream>>>(data, thread_arg->data, N, thread_arg->threadId);

    hipStreamSynchronize(0);

    float *host_data = (float *)(malloc(N * sizeof(float)));
    hipMemcpy(host_data, data, N*sizeof(float), hipMemcpyDeviceToHost);

    for(int i = 0; i < N; ++i)
    	std::cout << host_data[i] << " ";

    std::cout << std::endl;

    return NULL;
}


int main()
{
    const int num_threads = 4;

    pthread_t threads[num_threads];
    hipStream_t streams[num_threads];
    cuda_streams_arg stream_args[num_threads];

    float host_globalData[N];
    for (size_t idx = 0; idx < N; ++idx)
    {
    	host_globalData[idx] = idx*2;
    	std::cout << host_globalData[idx] << " ";
    }
    std::cout << std::endl;

    float *dev_globalData;
    hipMalloc(&dev_globalData, 10*sizeof(float));
    hipMemcpy(dev_globalData, host_globalData, 10*sizeof(float), hipMemcpyHostToDevice);


    for (int i = 0; i < num_threads; i++) {
    	stream_args[i].stream = &streams[i];
    	stream_args[i].data = dev_globalData;
    	stream_args[i].threadId = i;

    	if (pthread_create(&threads[i], NULL, launch_kernel, &stream_args[i])) {
            fprintf(stderr, "Error creating threadn");
            return 1;
        }
    }

    for (int i = 0; i < num_threads; i++) {
        if(pthread_join(threads[i], NULL)) {
            fprintf(stderr, "Error joining threadn");
            return 2;
        }
    }

    hipDeviceReset();

    return 0;
}
